#include "hip/hip_runtime.h"


#include <stdio.h>
#include <time.h>

#define N 4 
#define M (10000)
#define THREADS_PER_BLOCK 1024

bool checkPalindromoHost(char* s){
    for(int i = 0; i < N; i++){
        if(s[i] != s[N-i]){
            return false;
        }
    }
    return true;
}

__global__ void checkPalindromoDevice(char* s){
    bool b;
    for(int i = 0; i < N; i++){
        if(s[i] != s[N-i]){
            b= false;
        }
    }
    b= true;
}

int main()
{
	clock_t start,end;
    char *a;
    int size = N * sizeof( char );
    a = (char *)malloc( size );

    for (int  i = 0;i<N;i++){
        a[i] = (random() % 26);
    }

    /*a[0] = 'a';
    a[1] = 'v';
    a[2] = 'v';
    a[3] = 'a';*/

    start = clock();
	bool b = checkPalindromoHost(a);
    end = clock();

    float time1 = ((float)(end-start))/CLOCKS_PER_SEC;
    printf("CPU: %f seconds\n",time1);

    start = clock();

    char *p;

    hipMalloc( (void **) &p, size );
  
    hipMemcpy( p, a, size, hipMemcpyHostToDevice );
   
    bool pe;
    checkPalindromoDevice<<< (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( p);

    free(a);
   
    hipFree( p );  

    end = clock();
    float time2 = ((float)(end-start))/CLOCKS_PER_SEC;
    printf("CUDA: %f seconds, Speedup: %f\n",time2, time1/time2);

    return 0;
}
